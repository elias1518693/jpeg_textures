#include "hip/hip_runtime.h"
#define CUB_DISABLE_BF16_SUPPORT

// === required by GLM ===
#define GLM_FORCE_CUDA
#define CUDA_VERSION 12000
namespace std {
	using size_t = ::size_t;
};
// =======================

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>

#include "./libs/glm/glm/glm.hpp"
#include "./libs/glm/glm/gtc/matrix_transform.hpp"
#include "./libs/glm/glm/gtc/matrix_access.hpp"
#include "./libs/glm/glm/gtx/transform.hpp"
#include "./libs/glm/glm/gtc/quaternion.hpp"

#include "./utils.cuh"
#include "./BitReaderGPU.cuh"
#include "./HostDeviceInterface.h"
#include "./HashMap.cuh"
#include "./dct.cuh"

using glm::ivec2;
using glm::i8vec4;

constexpr uint32_t UV_BITS = 12;
constexpr float UV_FACTOR = 1 << UV_BITS;

struct Decoded {
	vec2 uv;
	int id;
};

struct HuffmanTable {
	int num_codes_per_bit_length[16];
	int huffman_values[256];
	int huffman_keys[256];
	int codelengths[256];
};

struct QuantizationTable {
	int values[64];
};

struct TextureData {
	int width;
	int height;
	uint8_t* data;
	HuffmanTable* huffmanTables;
	QuantizationTable* quanttables;
	uint32_t* mcuPositions;
};

#define RGBA(r, g, b) ((uint32_t(255) << 24) | (uint32_t(r) << 16) | (uint32_t(g) << 8) | uint32_t(b))

// retrieve the 4 texels around the given uv coordinate
void getTexels(
	uint32_t texID,
	vec2 uv, 
	TextureData* tex, 
	uint32_t* decoded,
	HashMap& decodedMcuMap,
	vec4* t00,
	vec4* t01,
	vec4* t10,
	vec4* t11
){
	float ftx = fmodf(uv.x, 1.0f) * float(tex->width);
	float fty = fmodf(uv.y, 1.0f) * float(tex->height);

	float ftlx = fmodf(ftx, 16.0f);
	float ftly = fmodf(fty, 16.0f);

	*t00 = {0.0f, 0.0f, 0.0f, 255.0f};
	*t01 = {0.0f, 0.0f, 0.0f, 255.0f};
	*t10 = {0.0f, 0.0f, 0.0f, 255.0f};
	*t11 = {0.0f, 0.0f, 0.0f, 0.0f};

	auto toVec4 = [](uint32_t color){
		return vec4{
			(color >>  0) & 0xff,
			(color >>  8) & 0xff,
			(color >> 16) & 0xff,
			(color >> 24) & 0xff,
		}; 
	};

	// EVALUATION: Clamp to a single MCU to check impact of accessing single vs. multiple MCUs
	// ftlx = clamp(ftlx, 0.6f, 15.4f);
	// ftly = clamp(ftly, 0.6f, 15.4f);
	// ftx = floor(ftx / 16.0f) * 16.0f + ftlx;
	// fty = floor(fty / 16.0f) * 16.0f + ftly;

	if(ftlx > 0.5f && ftlx < 15.5f && ftly > 0.5f && ftly < 15.5f){
		// Easy and fast case: All texels in same MCU

		int tx = ftx - 0.5f;
		int ty = fty - 0.5f;

		int mcu = tx / 16 + ty / 16 * tex->width / 16;
		uint32_t key = ((mcu & 0xffff) << 16) | (texID & 0xffff);

		uint32_t value;
		if(decodedMcuMap.get(key, &value)){
			uint32_t decodedMcuIndex = value & 0x00ffffff;
			
			tx %= 16;
			ty %= 16;
			int offset_00 = (tx + 0) % 8 + ((tx + 0) / 8) * 64 + ((ty + 0) % 8) * 8 + ((ty + 0) / 8) * 128;
			int offset_01 = (tx + 0) % 8 + ((tx + 0) / 8) * 64 + ((ty + 1) % 8) * 8 + ((ty + 1) / 8) * 128;
			int offset_10 = (tx + 1) % 8 + ((tx + 1) / 8) * 64 + ((ty + 0) % 8) * 8 + ((ty + 0) / 8) * 128;
			int offset_11 = (tx + 1) % 8 + ((tx + 1) / 8) * 64 + ((ty + 1) % 8) * 8 + ((ty + 1) / 8) * 128;

			*t00 = toVec4(decoded[decodedMcuIndex * 256 + offset_00]);
			*t01 = toVec4(decoded[decodedMcuIndex * 256 + offset_01]);
			*t10 = toVec4(decoded[decodedMcuIndex * 256 + offset_10]);
			*t11 = toVec4(decoded[decodedMcuIndex * 256 + offset_11]);

			// { // EVALUATION: Only for vr-caching figure. Set in comments when not in use. 
			// 	uint32_t flag = (value & 0xff000000) >> 24;
			// 	if(flag == 0xff){
			// 		*t00 = *t00 * 0.5f + 0.5f * vec4{255.0f, 0.0f, 0.0f, 255.0f};
			// 		*t01 = *t01 * 0.5f + 0.5f * vec4{255.0f, 0.0f, 0.0f, 255.0f};
			// 		*t10 = *t10 * 0.5f + 0.5f * vec4{255.0f, 0.0f, 0.0f, 255.0f};
			// 		*t11 = *t11 * 0.5f + 0.5f * vec4{255.0f, 0.0f, 0.0f, 255.0f};
			// 	}else{
			// 		*t00 = *t00 * 0.5f + 0.5f * vec4{0.0f, 255.0f, 0.0f, 255.0f};
			// 		*t01 = *t01 * 0.5f + 0.5f * vec4{0.0f, 255.0f, 0.0f, 255.0f};
			// 		*t10 = *t10 * 0.5f + 0.5f * vec4{0.0f, 255.0f, 0.0f, 255.0f};
			// 		*t11 = *t11 * 0.5f + 0.5f * vec4{0.0f, 255.0f, 0.0f, 255.0f};
			// 	}
			// }
		}
	}else{
		
		// Trickier case: texels reside in adjacent MCUs, which may or may not be available.
		uint32_t v_00, v_01, v_10, v_11 = 0;

		auto texelCoordToKey = [&](int tx, int ty){
			uint32_t mcu = tx / 16 + ty / 16 * tex->width / 16;
			uint32_t key = ((mcu & 0xffff) << 16) | (texID & 0xffff);
			return key;
		};

		bool v00Exists = decodedMcuMap.get(texelCoordToKey(ftx - 0.5f, fty - 0.5f), &v_00);
		bool v01Exists = decodedMcuMap.get(texelCoordToKey(ftx - 0.5f, fty + 0.5f), &v_01);
		bool v10Exists = decodedMcuMap.get(texelCoordToKey(ftx + 0.5f, fty - 0.5f), &v_10);
		bool v11Exists = decodedMcuMap.get(texelCoordToKey(ftx + 0.5f, fty + 0.5f), &v_11);

		auto toTexel = [&](uint32_t value, int tx, int ty){
			uint32_t decodedMcuIndex = value & 0x00ffffff;

			tx %= 16;
			ty %= 16;
			int offset = tx % 8 + (tx / 8) * 64 + (ty% 8) * 8 + (ty / 8) * 128;

			vec4 color = toVec4(decoded[decodedMcuIndex * 256 + offset]);

			// { // EVALUATION: Only for vr-caching figure. Set in comments when not in use. 
			// 	uint32_t flag = (value & 0xff000000) >> 24;
			// 	if(flag == 0xff){
			// 		return 0.5f * color + 0.5f * vec4{255.0f, 0.0f, 0.0f, 255.0f};
			// 	}else{
			// 		return 0.5f * color + 0.5f * vec4{0.0f, 255.0f, 0.0f, 255.0f};
			// 	}
			// }

			return color;
		};

		// If a texel's MCU is not decoded, clamp to one of the decoded MCUs
		
		// texel 00
		if(v00Exists)        *t00 = toTexel(v_00, ftx - 0.5f, fty - 0.5f);
		else if(v10Exists)   *t00 = toTexel(v_10, ftx + 0.5f, fty - 0.5f);
		else if(v01Exists)   *t00 = toTexel(v_01, ftx - 0.5f, fty + 0.5f);
		else if(v11Exists)   *t00 = toTexel(v_11, ftx + 0.5f, fty + 0.5f);
		
		// texel 01
		if(v01Exists)        *t01 = toTexel(v_01, ftx - 0.5f, fty + 0.5f);
		else if(v00Exists)   *t01 = toTexel(v_00, ftx - 0.5f, fty - 0.5f);
		else if(v10Exists)   *t01 = toTexel(v_10, ftx + 0.5f, fty - 0.5f);
		else if(v11Exists)   *t01 = toTexel(v_11, ftx + 0.5f, fty + 0.5f);

		// texel 10
		if(v10Exists)        *t10 = toTexel(v_10, ftx + 0.5f, fty - 0.5f);
		else if(v00Exists)   *t10 = toTexel(v_00, ftx - 0.5f, fty - 0.5f);
		else if(v01Exists)   *t10 = toTexel(v_01, ftx - 0.5f, fty + 0.5f);
		else if(v11Exists)   *t10 = toTexel(v_11, ftx + 0.5f, fty + 0.5f);

		// texel 11
		if(v11Exists)        *t11 = toTexel(v_11, ftx + 0.5f, fty + 0.5f);
		else if(v00Exists)   *t11 = toTexel(v_00, ftx - 0.5f, fty - 0.5f);
		else if(v01Exists)   *t11 = toTexel(v_01, ftx - 0.5f, fty + 0.5f);
		else if(v10Exists)   *t11 = toTexel(v_10, ftx + 0.5f, fty - 0.5f);
	}
}

uint32_t sampleColor_nearest(
	uint32_t texID,
	vec2 uv, 
	TextureData* texturesData, 
	uint32_t* decoded,
	HashMap& decodedMcuMap
){

	uint32_t color = 0;
	auto tex = &texturesData[texID];
	int tx = (int(uv.x * tex->width) % tex->width);
	int ty = (int(uv.y * tex->height) % tex->height);
	int mcu = tx / 16 + ty / 16 * tex->width / 16;
	uint32_t key = ((mcu & 0xffff) << 16) | (texID & 0xffff);

	uint32_t value;
	if(decodedMcuMap.get(key, &value)){
		uint32_t decodedMcuIndex = value & 0x00ffffff;
		bool isNewlyDecoded = (value >> 31) != 0;

		int tx = (int(uv.x * tex->width) % tex->width);
		int ty = (int(uv.y * tex->height) % tex->height);

		tx %= 16;
		ty %= 16;
		int offset = tx % 8 + (tx / 8) * 64 + (ty % 8) * 8 + (ty / 8) * 128;
		color = decoded[decodedMcuIndex * 256 + offset];

	}else{
		color = 0x00000000;
	}

	return color;
}

uint32_t sampleColor_linear(
	uint32_t texID,
	vec2 uv, 
	TextureData* texturesData, 
	uint32_t* decoded,
	HashMap& decodedMcuMap
){

	uint32_t color = 0xff000000;
	uint8_t* rgba = (uint8_t*)&color;
	auto tex = &texturesData[texID];

	// Apply 0.5f offset so that interpolated colors align with nearest-neighbor colors, i.e., the color sample is at the center.
	float ftx = fmodf(uv.x - 0.5f / float(tex->width), 1.0f) * float(tex->width);
	float fty = fmodf(uv.y - 0.5f / float(tex->height), 1.0f) * float(tex->height);

	vec4 t00, t01, t10, t11;
	getTexels(texID, uv, tex, decoded, decodedMcuMap, &t00, &t01, &t10, &t11);

	float wx = fmodf(ftx, 1.0f);
	float wy = fmodf(fty, 1.0f);

	vec4 interpolated = 
		(1.0f - wx) * (1.0f - wy) * t00 + 
		wx * (1.0f - wy) * t10 + 
		(1.0f - wx) * wy * t01 + 
		wx * wy * t11;

	rgba[0] = interpolated.r;
	rgba[1] = interpolated.g;
	rgba[2] = interpolated.b;
	rgba[3] = 255;

	return color;
}

void idct8x8_optimized(float* block, int thread) {
	auto cuda_block = cg::this_thread_block();

	cuda_block.sync();

	// Perform IDCT on rows
	CUDAsubroutineInplaceIDCTvector(&block[thread * 8], 1);

	cuda_block.sync();

	// Perform IDCT on columns
	CUDAsubroutineInplaceIDCTvector(&block[thread], 8);
}

int decodeHuffman(BitReaderGPU& bit_reader, const HuffmanTable& huffman_table) {
	int code = 0;
	int offset = 0;
	for (int bit_length = 1; bit_length <= 16; bit_length++) {
		code = (code << 1) | bit_reader.read_bit();
		int code_count = huffman_table.num_codes_per_bit_length[bit_length - 1];
#pragma unroll
		for (int j = 0; j < code_count; j++) {
			if (huffman_table.huffman_keys[offset + j] == code) {
				return huffman_table.huffman_values[offset + j];
			}
		}
		offset += huffman_table.num_codes_per_bit_length[bit_length - 1];
	}
	return -1;
}

int decodeHuffman_warpwide(BitReaderGPU& bit_reader, const HuffmanTable& huffman_table) {

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();
	auto warp = cg::tiled_partition<32>(block);
	auto index = grid.thread_rank();

	uint32_t code_peek = bit_reader.peek16Bit2();

	// decode up to 128 huffman codes. (4 iterations, 32 codes per iteration)
	// changed to 192 (6 iterations, 32 codes)
	#pragma unroll
	for(int i = 0; i < 6; i++){
		int codeIndex = i * 32 + warp.thread_rank();
		uint32_t bit_length = huffman_table.codelengths[codeIndex];
		uint32_t code = code_peek >> (16 - bit_length);
		bool isValidCode = huffman_table.huffman_keys[codeIndex] == code;

		uint32_t mask = warp.ballot(isValidCode);

		if(mask > 0){
			int winningLane = __ffs(mask) - 1;

			// the winning lane broadcasts the huffman value and the bit length to all other threads
			bit_length = warp.shfl(bit_length, winningLane);
			uint32_t huffmanValue = warp.shfl(huffman_table.huffman_values[codeIndex], winningLane);

			bit_reader.advance(bit_length);

			return huffmanValue;
		}
	}

	return -1;
}

int DecodeNumber(int code, int bits) {
	int l = 1 << (code - 1);
	if (bits >= l) {
		return bits;
	}
	else {
		return bits - (2 * l - 1);
	}
}

void decodeCoefficients(BitReaderGPU& bit_reader, HuffmanTable& huffman_table, float* coefficients, int previous_dc) {
	coefficients[0] = previous_dc;
	int i = 1;
#pragma unroll
	while (i < 64) {
		int ac_code = decodeHuffman(bit_reader, huffman_table);

		if (ac_code == 0) {
			break;
		}
		int size = ac_code;
		if (ac_code > 15) {
			int run_length = (ac_code >> 4) & 0xF;
			size = ac_code & 0xF;
			i += run_length;
		}
		if (i >= 64) break;
		int ac_value = bit_reader.read_bits(size);
		ac_value = DecodeNumber(size, ac_value);
		coefficients[i++] = ac_value;
	}
}

void decodeCoefficients_warpwide(
	BitReaderGPU& bit_reader,
	const HuffmanTable& huffman_table,
	float* sh_coefficients
) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();
	auto warp = cg::tiled_partition<32>(block);
	int i = 1;
#pragma unroll
	while (i < 64) {
		int ac_code = decodeHuffman_warpwide(bit_reader, huffman_table);

		if (ac_code == 0) {
			break;
		}
		int size = ac_code;
		if (ac_code > 15) {
			int run_length = (ac_code >> 4) & 0xF;
			size = ac_code & 0xF;
			i += run_length;
		}

		if (i >= 64) break;

		int ac_value = bit_reader.read_bits(size);
		ac_value = DecodeNumber(size, ac_value);

		sh_coefficients[i++] = ac_value;

		// warp.sync();
	}
}


__constant__ int dezigzag_order[64] = {
	0, 1, 8, 16, 9, 2, 3, 10,
 17, 24, 32, 25, 18, 11, 4, 5,
 12, 19, 26, 33, 40, 48, 41, 34,
 27, 20, 13, 6, 7, 14, 21, 28,
 35, 42, 49, 56, 57, 50, 43, 36,
 29, 22, 15, 23, 30, 37, 44, 51,
 58, 59, 52, 45, 38, 31, 39, 46,
 53, 60, 61, 54, 47, 55, 62, 63
};

// fetch bit-offset to the mcu from the indexing table
int calculate_datastart(int mcu, const uint32_t* mcu_index) {
	int packed_index = (mcu / 9) * 5;
	int offset_within_packed = mcu % 9;
	int absolute_offset = mcu_index[packed_index];
	if (offset_within_packed == 0) {
		return absolute_offset;
	}

	int rel_index = offset_within_packed - 1; 
	int word = mcu_index[packed_index + 1 + (rel_index / 2)];
	int shift = (rel_index % 2 == 0) ? 16 : 0;
	int relative_offset = (word >> shift) & 0xFFFF;
	return absolute_offset + relative_offset;
}

uint16_t get12bit(const uint8_t* buf, int idx) {
	int group = idx / 2;
	int byte_idx = group * 3;

	if ((idx % 2) == 0) {
		return (buf[byte_idx]) | ((buf[byte_idx + 1] & 0x0F) << 8);
	}
	else {
		return ((buf[byte_idx + 1] >> 4) & 0x0F) | (buf[byte_idx + 2] << 4);
	}
}

uint32_t uvToMCUIndex(int width, int height, float u, float v) {
	int tx = (int(u * width) % width);
	int ty = (int(v * height) % height);
	return tx / 16 + ty / 16 * width / 16;
}

extern "C" __global__
void kernel_mark(
	uint32_t width, 
	uint32_t height,
	hipSurfaceObject_t gl_uvs,
	hipSurfaceObject_t gl_miplevel,
	uint32_t* toDecode,
	uint32_t* toDecodeCounter,
	TextureData* texturesData,
	int num_textures,
	HashMap decodedMcuMap
) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	// int x = grid.block_index().x * 8 + block.thread_index().x;
	// int y = grid.block_index().y * 8 + block.thread_index().y;

	int pixelID = grid.thread_rank();
	int x = pixelID % width;
	int y = pixelID / width;

	if(x >= width) return;
	if(y >= height) return;


	uint32_t pixelVal;
	surf2Dread(&pixelVal, gl_uvs, x * 4, y);
	uint32_t uvX = (pixelVal >>  0) & 0xffff;
	uint32_t uvY = (pixelVal >> 16) & 0xffff;
	vec2 uv = {float(uvX) / 65536.0f, float(uvY) / 65536.0f};

	uint32_t pixelMiplevel;
	surf2Dread(&pixelMiplevel, gl_miplevel, x * 4, y);
	uint32_t mipLevel = min(float((pixelMiplevel >> 0) & 0xffff) / 256.0f, 7.0);
	uint32_t texID = (pixelMiplevel >> 16) & 0xffff;
	uint32_t texID_mipmap = texID * 8 + mipLevel;

	if (texID >= num_textures) return;
	
	// key: mcu's are identified by their index, texture-id, and mip map level. 
	uint32_t mcu = uvToMCUIndex(texturesData[texID_mipmap].width, texturesData[texID_mipmap].height, uv.x, uv.y);
	uint32_t key = ((mcu & 0xffff) << 16) | (texID_mipmap & 0xffff);

	// to avoid contention, make sure that for any MCU, only one thread per warp continues.
	{ 
		// mask of warp threads with same key
		auto block = cg::this_thread_block();
		auto warp = cg::tiled_partition<32>(block);
		uint32_t mask = warp.match_any(key);

		// find the lowest lane between threads with the same key
		int winningLane = __ffs(mask) - 1;

		// return early because another thread handles this MCU
		if(warp.thread_rank() != winningLane) return;
	}

	// Reserve a spot in the hash map. The value, the TB-Slot, will be acquired and set by the decode kernel
	bool alreadyExists = false;
	int location = 0;
	decodedMcuMap.set(key, 0, &location, &alreadyExists);

	if(!alreadyExists){
		// Add MCU to decoder queue
		uint32_t decodeIndex = atomicAdd(toDecodeCounter, 1);
		toDecode[decodeIndex] = key;
	}else{
		// MCU is in cache - flag as visible
		atomicOr(&decodedMcuMap.entries[location], 0x00000000'ff000000);
	}

}

extern "C" __global__
void kernel_decode_420(
	uint32_t* toDecode,
	uint32_t* decoded,
	TextureData* texturesData,
	HashMap decodedMcuMap,
	uint32_t* TBSlots,
	uint32_t firstAvailableTBSlotsIndex
) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();
	auto warp = cg::tiled_partition<32>(block);
	int thread = block.thread_rank();

	uint32_t textureInfo = toDecode[grid.block_rank()];
	uint32_t texture_id = ((textureInfo >> 0) & 0xffff);
	uint32_t mcu = (textureInfo >> 16) & 0xffff;

	block.sync();

	__shared__ float sh_coefficients[384];
	__shared__ float sh_dezigzagged[510];
	__shared__ uint8_t sh_data[510];

	const TextureData& textureData = texturesData[texture_id];

	int datastart = calculate_datastart(mcu, textureData.mcuPositions);
	int datastartbit = datastart % 8;

	// Load compressed data into shared memory
	for (int i = 0; i < 6; i++) {
		sh_data[block.thread_rank() + 64 * i] = textureData.data[datastart / 8 + block.thread_rank() + 64 * i];
		sh_coefficients[block.thread_rank() + 64 * i] = 0;
	}

	block.sync();

	BitReaderGPU bit_reader(&sh_data[0], datastartbit);

	// Decode DC's with first warp
	if (warp.meta_group_rank() == 0)
	{
		int previousDC = (bit_reader.read_bits(12) & 0x0fff) - 2048;
		sh_coefficients[0] = previousDC;
		for (int i = 1; i < 4; i++) {

			int huff_index = (i <= 3) ? 0 : (i - 3);
			const HuffmanTable& huffmanTable = textureData.huffmanTables[huff_index];
			int dc_value = decodeHuffman_warpwide(bit_reader, huffmanTable);
			if (dc_value > 0) {
				int dc_difference = bit_reader.read_bits(dc_value);
				dc_value = DecodeNumber(dc_value, dc_difference);
			}

			sh_coefficients[i * 64] = dc_value + previousDC;
			previousDC = dc_value + previousDC;
		}
		sh_coefficients[64 * 4] = (bit_reader.read_bits(12) & 0x0fff) - 2048;
		sh_coefficients[64 * 5] = (bit_reader.read_bits(12) & 0x0fff) - 2048;
	}

	block.sync();

	// Decode AC's with first warp
	if (warp.meta_group_rank() == 0)
	{
		for (int i = 0; i < 6; i++) {
			int huff_index = (i <= 3) ? 0 : (i - 3);

			const HuffmanTable& huffmanTable = textureData.huffmanTables[3 + huff_index];
			int previousDC = 0;

			decodeCoefficients_warpwide(bit_reader, huffmanTable, &sh_coefficients[i * 64]);
		}
	}

	block.sync();

	QuantizationTable* quanttable1 = &textureData.quanttables[0];
	QuantizationTable* quanttable2 = &textureData.quanttables[1];
	for (int i = 0; i < 4; i++)
		sh_dezigzagged[dezigzag_order[thread] + i * 64] = sh_coefficients[threadIdx.x + 64 * i] * quanttable1->values[thread];
	for (int i = 0; i < 2; i++)
		sh_dezigzagged[dezigzag_order[thread] + i * 64 + 256] = sh_coefficients[threadIdx.x + 256 + 64 * i] * quanttable2->values[thread];

	block.sync();

	idct8x8_optimized(&sh_dezigzagged[(thread / 8) * 64], thread % 8);

	block.sync();

	// Acquire a texture block cache slot
	__shared__ int sh_tbslot;
	if(block.thread_rank() == 0){
		uint32_t slotIndex = firstAvailableTBSlotsIndex + grid.block_rank();
		uint32_t tbslot = TBSlots[slotIndex];
		uint32_t visFlag = 0b0000'0001; // mark as visible & newly cached. 
		uint32_t value = (visFlag << 24) | tbslot;
		uint64_t entry = (uint64_t(textureInfo) << 32) | uint64_t(value);
		
		bool alreadyExists = false;
		int location = 0;
		decodedMcuMap.set(textureInfo, 0, &location, &alreadyExists);
		atomicExch(&decodedMcuMap.entries[location], entry);

		sh_tbslot = tbslot;
	}

	block.sync();

	// Write decoded texels to texture block cache
	for (int i = 0; i < 4; i++) {
		uint8_t* rgba = (uint8_t*)&decoded[sh_tbslot * 256 + threadIdx.x + 64 * i];
		float y = sh_dezigzagged[threadIdx.x + 64 * i] + 128.0f;

		int chroma_x = (thread % 8) / 2;
		int chroma_y = (thread / 8) / 2;
		int chroma_index = chroma_y * 8 + chroma_x + i / 2 * 4 * 8 + i % 2 * 4;

		float cb = sh_dezigzagged[chroma_index + 64 * 4];
		float cr = sh_dezigzagged[chroma_index + 64 * 5];

		rgba[0] = clamp(y + 1.402f * cr, 0.0f, 255.0f);
		rgba[1] = clamp(y - 0.344136f * cb - 0.714136f * cr, 0.0f, 255.0f);
		rgba[2] = clamp(y + 1.772f * cb, 0.0f, 255.0f);
		rgba[3] = 255;
	}
}


// This kernel is used to indirectly launch kernel_decode_420 from the GPU, 
// so that we don't have to memcpy <toDecodeCounter> to host before launching it. 
extern "C" __global__
void kernel_launch_decode(
	uint32_t* toDecodeCounter,
	uint32_t* TBSlots,
	uint32_t* TBSlotsCounter,
	uint32_t* toDecode,
	uint32_t* decoded,
	TextureData* texturesData,
	HashMap decodedMcuMap
) {
	auto grid = cg::this_grid();

	if(grid.thread_rank() == 0){

		int numBlocks = *toDecodeCounter;
		uint32_t firstAvailableTBSlotsIndex = *TBSlotsCounter;

		kernel_decode_420<<<numBlocks, 64>>>(
			toDecode,
			decoded,
			texturesData,
			decodedMcuMap,
			TBSlots,
			*TBSlotsCounter
		);

		*TBSlotsCounter = (*toDecodeCounter) + (*TBSlotsCounter);
	}
}

extern "C" __global__
void kernel_resolve(
	CommonLaunchArgs args,
	uint32_t viewIndex,
	uint32_t width, 
	uint32_t height,
	hipSurfaceObject_t gl_desktop,
	// hipSurfaceObject_t gl_desktop_depth,
	hipSurfaceObject_t gl_uvs,
	hipSurfaceObject_t gl_miplevel,
	bool showUVs,
	uint32_t* toDecode,
	uint32_t* decoded,
	TextureData* texturesData,
	int num_textures,
	HashMap decodedMcuMap
) {

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	int x = grid.block_index().x * 8 + block.thread_index().x;
	int y = grid.block_index().y * 8 + block.thread_index().y;

	if(x >= width) return;
	if(y >= height) return;

	uint32_t pixelUVs;
	surf2Dread(&pixelUVs, gl_uvs, x * 4, y);

	uint32_t uvX = (pixelUVs >>  0) & 0xffff;
	uint32_t uvY = (pixelUVs >> 16) & 0xffff;
	vec2 uv = {float(uvX) / 65536.0f, float(uvY) / 65536.0f};

	uint32_t pixelMiplevel;
	surf2Dread(&pixelMiplevel, gl_miplevel, x * 4, y);
	uint32_t mipLevel = min(float((pixelMiplevel >> 0) & 0xffff) / 256.0f, 7.0);
	uint32_t texID = (pixelMiplevel >> 16) & 0xffff;
	uint32_t texID_mipmap = texID * 8 + mipLevel;
	if (pixelUVs == 0) return;

	uint32_t color = 0;
	uint8_t* rgba = (uint8_t*)&color;

	if(args.uniforms.enableLinearInterpolation){
		color = sampleColor_linear(texID_mipmap, uv, texturesData, decoded, decodedMcuMap);
	}else{
		color = sampleColor_nearest(texID_mipmap, uv, texturesData, decoded, decodedMcuMap);
	}

	// if(viewIndex > 0)
	if(args.uniforms.showCaching){
		auto tex = &texturesData[texID_mipmap];
		int tx = (int(uv.x * tex->width) % tex->width);
		int ty = (int(uv.y * tex->height) % tex->height);
		int mcu = tx / 16 + ty / 16 * tex->width / 16;
		uint32_t key = ((mcu & 0xffff) << 16) | (texID_mipmap & 0xffff);
		uint32_t value;
		if(decodedMcuMap.get(key, &value)){

			uint32_t decodedMcuIndex = value & 0x00ffffff;
			bool isNewlyDecoded = (value >> 24) == 0b00000001;

			if(isNewlyDecoded){
				// color = 0xff0000ff;
				rgba[0] = float(rgba[0]) * 0.5f + 125.0f;
				rgba[1] = float(rgba[1]) * 0.5f + 0.0f;
				rgba[2] = float(rgba[2]) * 0.5f + 0.0f;
			}else{
				// color = 0xff00ff00;
				rgba[0] = float(rgba[0]) * 0.5f + 0.0f;
				rgba[1] = float(rgba[1]) * 0.5f + 125.0f;
				rgba[2] = float(rgba[2]) * 0.5f + 0.0f;
			}
		}
	}else if(args.uniforms.showUVs){
		rgba[0] = 256.0f * uv.x;
		rgba[1] = 256.0f * uv.y;
		rgba[2] = 0;
	}else if(args.uniforms.showMCUs){
		auto tex = &texturesData[texID_mipmap];
		int tx = (int(uv.x * tex->width) % tex->width);
		int ty = (int(uv.y * tex->height) % tex->height);
		int mcu = tx / 16 + ty / 16 * tex->width / 16;
		color = mcu * 123456;

		rgba[0] = (texID * 1234) % 256;

		color = mipLevel * 12345;
		color = mcu * texID * mipLevel * 12345;


		int level = float(mipLevel) * 14.0f / 7.0f;
		if(level == 10) color = 0xff42019e;
		if(level ==  9) color = 0xff4f3ed5;
		if(level ==  8) color = 0xff436df4;
		if(level ==  7) color = 0xff61aefd;
		if(level ==  6) color = 0xff8be0fe;
		if(level ==  5) color = 0xffbfffff;
		if(level ==  4) color = 0xff98f5e6;
		if(level ==  3) color = 0xffa4ddab;
		if(level ==  2) color = 0xffa5c266;
		if(level ==  1) color = 0xffbd8832;
		if(level ==  0) color = 0xffa24f5e;

		float wMcu = 0.0;
		if((tx / 16 + ty / 16) % 2 == 0){
			wMcu = 1.0f;
		}else{
			wMcu = 0.8f;
		}

		uint32_t c2 = texID * 12345678901898;
		uint8_t* rgbac2 = (uint8_t*)&c2;


		rgba[0] = clamp((0.5f * float(rgba[0]) + 0.5f * float(rgbac2[0])) * wMcu, 0.0f, 255.0f);
		rgba[1] = clamp((0.5f * float(rgba[1]) + 0.5f * float(rgbac2[1])) * wMcu, 0.0f, 255.0f);
		rgba[2] = clamp((0.5f * float(rgba[2]) + 0.5f * float(rgbac2[2])) * wMcu, 0.0f, 255.0f);
	}else if(args.uniforms.showTexID){
		color = texID * 123456;
	}else if(args.uniforms.showMipLevel){
		int level = float(mipLevel) * 14.0f / 7.0f;
		if(level == 10) color = 0xff42019e;
		if(level ==  9) color = 0xff4f3ed5;
		if(level ==  8) color = 0xff436df4;
		if(level ==  7) color = 0xff61aefd;
		if(level ==  6) color = 0xff8be0fe;
		if(level ==  5) color = 0xffbfffff;
		if(level ==  4) color = 0xff98f5e6;
		if(level ==  3) color = 0xffa4ddab;
		if(level ==  2) color = 0xffa5c266;
		if(level ==  1) color = 0xffbd8832;
		if(level ==  0) color = 0xffa24f5e;
	}

	// color = texID * 12345;
	color = color | 0xff000000;

	// x = clamp(x, 0, width - 1);
	// y = clamp(y, 0, height - 1);

	// if(x == 0 && y == 0) printf("width: %d, height: %d \n", width, height);
	// color = 0xff0000ff;

	surf2Dwrite(color, gl_desktop, x * 4, y);
}


extern "C" __global__
void kernel_init_availableMcuSlots(
	uint32_t* TBSlots,
	uint32_t* TBSlotsCounter,
	uint32_t numDecodedMcuCapacity
) {

	auto grid = cg::this_grid();

	if(grid.thread_rank() >= numDecodedMcuCapacity) return;

	TBSlots[grid.thread_rank()] = grid.thread_rank();
}

extern "C" __global__
void kernel_update_cache(
	HashMap decodedMcuMap_source,
	HashMap decodedMcuMap_target,
	uint32_t* TBSlots,
	uint32_t* TBSlotsCounter,
	bool freezeCache
) {

	auto grid = cg::this_grid();

	if(grid.thread_rank() >= decodedMcuMap_source.capacity) return;

	uint64_t entry = decodedMcuMap_source.entries[grid.thread_rank()];
	uint32_t key = entry >> 32;
	uint32_t value = entry & 0xffffffff;
	uint32_t visFlag  = (value >> 24) & 0xff;
	uint32_t slot = (value >>  0)  & 0xffffff;

	bool isMcuVisible = visFlag != 0;
	bool isNewlyDecoded = (visFlag == 0b00000001);

	// Note: If cache is frozen: Only remove newly decoded entries 
	// but preserve previously cached entries, including those that are currently invisible. 

	if(entry == HashMap::EMPTYENTRY) return;

	// Put slot to decoded texture block back in pool of slots
	auto remove = [&](){
		uint32_t old = atomicSub(TBSlotsCounter, 1);
		int slotIndex = int(old) - 1;

		TBSlots[slotIndex] = slot;
	};

	// Replicate entry in new hash map, which will be used in the next frame. 
	auto preserve = [&](){
		int location;
		bool alreadyExists;
		uint32_t newVal = (0x00 << 24) | slot;
		decodedMcuMap_target.set(key, newVal, &location, &alreadyExists);
	};

	if(freezeCache){
		if(isNewlyDecoded){
			remove();
		}else{
			preserve();
		}
	}else{
		if(isMcuVisible == 0){
			remove();
		}else{
			preserve();
		}
	}
}



extern "C" __global__
void kernel_eval_showCacheVR(
	uint32_t* toDecode,
	HashMap decodedMcuMap,
	uint32_t first,
	uint32_t count
) {
	auto grid = cg::this_grid();

	// if(grid.thread_rank() == 0){
	// 	// printf("test");
	// 	printf("%d \n", first);
	// }

	// return;

	if(grid.thread_rank() >= count) return;

	uint32_t i = first + grid.thread_rank();

	uint32_t key = toDecode[i];

	uint32_t value = 0;
	int location = 0;
	bool exists = decodedMcuMap.get(key, &value, &location);

	// if(i == 48'000){
	// 	// printf("[dbg]: key: %u, location: %d, value: %u \n", key, location, value);
	// 	// printf("decodedMcuMap.entries[%u] = %u | %u; \n", location, key, newValue);
	// }

	// if(grid.thread_rank() == 0) printf("i: %u \n", i);

	if(exists){
		// uint64_t flag = 0b1111'1111;
		uint64_t flag = 0xff;
		uint64_t slot = value & 0x00ffffff;

		uint64_t newValue = (flag << 24) | slot;
		uint64_t entry = (uint64_t(key) << 32) | newValue;
		decodedMcuMap.entries[location] = entry;
	}
}

extern "C" __global__
void kernel_dummy(
	uint32_t* var
) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	if(grid.thread_rank() != 0) return;

	*var = 123;
}